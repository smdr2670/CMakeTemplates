#include "hip/hip_runtime.h"
#include "myKernel.h"

// running parallel for each thread
__global__ 
void vectorAdd(int* a, int* b, int* c, int n){
    
    int i = threadIdx.x;
    if (i < n){
        c[i] = a[i] + b[i];
    }
}

// Wrapper function so the .cpp file has not to invoke the kernel directly
void myGPUvectorAdd(const dim3 gridSize, const dim3 blockSize, int* a, int* b, int* c, int n) {
    vectorAdd <<< gridSize, blockSize  >>> (a, b, c, n);
}

